
#include <hip/hip_runtime.h>
#include <iostream>

/// A kernel function designed to calculate the number of
///  numbers divisible by two, three, and five
///
/// # Arguments
/// * `d_number_of_divisible_by_two` - The number of numbers divisible by two
/// * `d_number_of_divisible_by_three` - The number of numbers divisible by three
/// * `d_number_of_divisible_by_five` - The number of numbers divisible by five
__global__ void calculate(
    unsigned long long int * d_number_of_divisible_by_two,
    unsigned long long int * d_number_of_divisible_by_three,
    unsigned long long int * d_number_of_divisible_by_five
) {
    int grid_x = blockIdx.x;
    int grid_y = blockIdx.y;
    int grid_z = blockIdx.z;

    int block_x = threadIdx.x;
    int block_y = threadIdx.y;
    int block_z = threadIdx.z;

    unsigned long long local_counter = 
        (grid_z * 100 * 100 * 10 * 10 * 10) + 
        (grid_y * 100 * 10 * 10) + 
        (grid_x * 10 * 10) +
        (block_z * 10 * 10) +
        (block_y * 10) +
        block_x + 1;

    unsigned long one = 1;

    if (local_counter % 2 == 0) {
        atomicAdd(d_number_of_divisible_by_two, one);
    }
    if (local_counter % 3 == 0) {
        atomicAdd(d_number_of_divisible_by_three, one);
    }
    if (local_counter % 5 == 0) {
        atomicAdd(d_number_of_divisible_by_five, one);
    }
}

int main() {
    // Say hello to the user
    std::cout << "Hello, Metis!" << std::endl;

    // Host variables
    unsigned long long int h_number_of_divisible_by_two   = 0;
    unsigned long long int h_number_of_divisible_by_three = 0;
    unsigned long long int h_number_of_divisible_by_five  = 0;

    // Device variables
    unsigned long long int * d_number_of_divisible_by_two;
    unsigned long long int * d_number_of_divisible_by_three;
    unsigned long long int * d_number_of_divisible_by_five;

    // Allocate memory on the device with the correct sizing
    hipMalloc( &d_number_of_divisible_by_two,   sizeof(unsigned long long int) );
    hipMalloc( &d_number_of_divisible_by_three, sizeof(unsigned long long int) );
    hipMalloc( &d_number_of_divisible_by_five,  sizeof(unsigned long long int) );

    // Copy the memory from the host to the device
    hipMemcpy( d_number_of_divisible_by_two,   &h_number_of_divisible_by_two,   
        sizeof(unsigned long long int), hipMemcpyHostToDevice );
    hipMemcpy( d_number_of_divisible_by_three, &h_number_of_divisible_by_three,
        sizeof(unsigned long long int), hipMemcpyHostToDevice );
    hipMemcpy( d_number_of_divisible_by_five,  &h_number_of_divisible_by_five,
        sizeof(unsigned long long int), hipMemcpyHostToDevice );

    // Define our grid's dimensions
    dim3 gridDim(100, 100, 10);

    // Define each block's dimensions
    dim3 blockDim(10, 10, 10);

    // Run our calculation
    calculate<<<gridDim, blockDim>>>(d_number_of_divisible_by_two, d_number_of_divisible_by_three, d_number_of_divisible_by_five);
    hipDeviceSynchronize();

    // Copy the memory back to our machine
    hipMemcpy(&h_number_of_divisible_by_two, d_number_of_divisible_by_two, sizeof(unsigned long long int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_number_of_divisible_by_three, d_number_of_divisible_by_three, sizeof(unsigned long long int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_number_of_divisible_by_five, d_number_of_divisible_by_five, sizeof(unsigned long long int), hipMemcpyDeviceToHost);

    // Provide our results to the user
    std::cout << std::endl
              << "- Numbers divisible by two: "       << h_number_of_divisible_by_two       << std::endl
              << "- Numbers divisible by three: "     << h_number_of_divisible_by_three     << std::endl
              << "- Numbers divisible by five: "      << h_number_of_divisible_by_five      << std::endl;

    // Free the memory
    hipFree(d_number_of_divisible_by_two);
    hipFree(d_number_of_divisible_by_three);
    hipFree(d_number_of_divisible_by_five);

    return 0;
}